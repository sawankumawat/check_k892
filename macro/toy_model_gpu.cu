#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "TH1F.h"
#include "TCanvas.h"
#include "TFile.h"
#include "TLorentzVector.h"

// constants
__constant__ double m_mother = 1.713;   // f0(1710) mass
__constant__ double m_daughter1 = 0.493; // Ks
__constant__ double m_daughter2 = 0.493; // Ks

// GPU kernel: generate events
__global__ void decayKernel(int nEvents, float *pT_array, float *recPt_array, unsigned long seed) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nEvents) return;

    // random state
    hiprandState state;
    hiprand_init(seed, i, 0, &state);

    // --- generate random kinematics ---
    float pT  = 20.0f * hiprand_uniform(&state);                    // pT in [0,20]
    float phi = (2.0f * M_PI) * (hiprand_uniform(&state) - 0.5f);   // phi in [-pi,pi]
    float eta = -0.8f + 1.6f * hiprand_uniform(&state);             // eta in [-0.8,0.8]

    // mother 4-momentum in lab frame (approximate: using TLorentzVector math on CPU side normally)
    float px = pT * cosf(phi);
    float py = pT * sinf(phi);
    float pz = pT * sinhf(eta);
    float E  = sqrtf(px*px + py*py + pz*pz + m_mother*m_mother);

    // store mother pT
    pT_array[i] = pT;

    // --- simple two-body decay in mother rest frame ---
    float M  = m_mother;
    float m1 = m_daughter1;
    float m2 = m_daughter2;

    // momentum of daughters in CM frame
    float p_star = sqrtf((M*M - (m1+m2)*(m1+m2))*(M*M - (m1-m2)*(m1-m2))) / (2*M);

    // pick random isotropic direction
    float costheta = 2.0f*hiprand_uniform(&state) - 1.0f;
    float sintheta = sqrtf(1.0f - costheta*costheta);
    float phi_decay = 2.0f*M_PI*hiprand_uniform(&state);

    float px1 = p_star * sintheta * cosf(phi_decay);
    float py1 = p_star * sintheta * sinf(phi_decay);
    float pz1 = p_star * costheta;
    float E1  = sqrtf(p_star*p_star + m1*m1);

    // reconstruct mother from daughters (back in CM → trivial)
    float rec_px = px1 - px1;  // should cancel, placeholder
    float rec_py = py1 - py1;
    float rec_pz = pz1 - pz1;
    float rec_E  = E1 + sqrtf(p_star*p_star + m2*m2);

    float rec_pT = sqrtf(rec_px*rec_px + rec_py*rec_py);
    recPt_array[i] = rec_pT;
}

int main() {
    int nEvents = 1e8;

    // allocate GPU arrays
    float *d_pT, *d_recPt;
    hipMalloc(&d_pT, nEvents*sizeof(float));
    hipMalloc(&d_recPt, nEvents*sizeof(float));

    // launch kernel
    int threads = 256;
    int blocks  = (nEvents + threads - 1) / threads;
    decayKernel<<<blocks, threads>>>(nEvents, d_pT, d_recPt, 1234UL);
    hipDeviceSynchronize();

    // copy back to CPU
    float *h_pT = new float[nEvents];
    float *h_recPt = new float[nEvents];
    hipMemcpy(h_pT, d_pT, nEvents*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_recPt, d_recPt, nEvents*sizeof(float), hipMemcpyDeviceToHost);

    // ROOT histograms
    TH1F *h1 = new TH1F("h1","Mother p_{T};p_{T} (GeV/c);Events",150,0,30);
    TH1F *h2 = new TH1F("h2","Reconstructed p_{T};p_{T} (GeV/c);Events",150,0,30);

    for (int i=0;i<nEvents;i++) {
        h1->Fill(h_pT[i]);
        h2->Fill(h_recPt[i]);
    }

    // draw and save
    TCanvas *c1 = new TCanvas("c1","GPU Hist",800,600);
    h1->SetLineColor(kBlue); h1->Draw();
    c1->SaveAs("mother_pT_gpu.png");

    TCanvas *c2 = new TCanvas("c2","GPU Reco",800,600);
    h2->SetLineColor(kRed); h2->Draw();
    c2->SaveAs("reconstructed_pT_gpu.png");

    // cleanup
    delete[] h_pT;
    delete[] h_recPt;
    hipFree(d_pT);
    hipFree(d_recPt);

    return 0;
}
